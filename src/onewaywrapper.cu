#include "hip/hip_runtime.h"
/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */
#include "utils.hpp"
#include "gpuHeaders.cuh"
#include <iostream>

using namespace std;

#define threadsPerBlock 256

int one_way_handshake(GraphData graph, int *& matches, int numthreads)
{
	int num_thread_blocks = (numthreads + threadsPerBlock - 1) / threadsPerBlock;
	
	int numVertices = graph.numNodes;
	int numEdges = graph.numEdges;
	
	//Prepare various GPU arrays that we're going to need:
	
	int * strongNeighbor_gpu;//will hold strongest neighbor for each vertex
	hipMalloc((void **)&strongNeighbor_gpu, numVertices * sizeof(int));
	hipMemcpy(strongNeighbor_gpu, matches, numVertices * sizeof(int), hipMemcpyHostToDevice);//init to all -1
	
	int * matches_gpu;//will hold the output
	hipMalloc((void **)&matches_gpu, numVertices * sizeof(int));
	hipMemcpy(matches_gpu, matches, numVertices * sizeof(int), hipMemcpyHostToDevice);
	
	int * src_gpu;//holds the src nodes in edge list
	int * dst_gpu;//holds the dst nodes in edge list
	int * weight_gpu;//holds the edge weights in edge list
	hipMalloc((void **)&src_gpu, (1+numEdges) * sizeof(int));
	hipMalloc((void **)&dst_gpu, (1+numEdges) * sizeof(int));
	hipMalloc((void **)&weight_gpu, (1+numEdges) * sizeof(int));
	hipMemcpy(src_gpu, graph.src, numEdges * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dst_gpu, graph.dst, numEdges * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(weight_gpu, graph.weight, numEdges * sizeof(int), hipMemcpyHostToDevice);
	
	int * temp1_gpu;//a temporary array for data we don't need to keep for long
	int * temp2_gpu;//a temporary array for data we don't need to keep for long
	int * temp3_gpu;//a temporary array for data we don't need to keep for long
	int * temp4_gpu;//a temporary array for data we don't need to keep for long
	hipMalloc((void **)&temp1_gpu, (1+numEdges) * sizeof(int));
	hipMalloc((void **)&temp2_gpu, (1+numEdges) * sizeof(int));
	hipMalloc((void **)&temp3_gpu, (1+numEdges) * sizeof(int));
	hipMalloc((void **)&temp4_gpu, (1+numEdges) * sizeof(int));
	
	int * madeChanges_gpu; //1-element array that strongestNeighborScan_gpu will mark to indicate whether it made changes
	hipMalloc((void **)&madeChanges_gpu, sizeof(int));
	
    /* Start matching */
    int iter;
    for (iter = 0; ; iter++) {
		
		//Step 1: Get strongest neighbor for each vertex/node
		
		//Step 1a: prepare initial values for segment scan:
		hipMemcpy(temp1_gpu, dst_gpu, numEdges * sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(temp3_gpu, weight_gpu, numEdges * sizeof(int), hipMemcpyDeviceToDevice);
		
		//Step 1b: segment scan; each vertex extends a hand to its strongest neighbor
		int distance = 1;
		while(true) {
			if(distance > numEdges * 2 + 1) {
				cerr << "ERROR: failed to stop segment scan on-time.\n";
				break;
			}
			
			hipMemset(madeChanges_gpu, 0, sizeof(int));
			strongestNeighborScan_gpu<<<num_thread_blocks, threadsPerBlock>>>(src_gpu, temp1_gpu, temp2_gpu, temp3_gpu, temp4_gpu, madeChanges_gpu, distance, numEdges);
			swapArray((void**) &temp1_gpu, (void**) &temp2_gpu);
			swapArray((void**) &temp3_gpu, (void**) &temp4_gpu);
			
			//break from segment scan if it's no longer doing anything
			int madeChanges = 0;
			hipMemcpy(&madeChanges, madeChanges_gpu, sizeof(int), hipMemcpyDeviceToHost);
			if(madeChanges == 0) {
				break;
			}
			
			distance *= 2;
		}
		int * strongestDst_gpu = temp1_gpu;
		temp1_gpu = NULL;
		//int * strongestWeight_gpu = temp3_gpu;
		
		//Step 1c: Collate strongest neighbors into strongNeighbor array
		collateSegments_gpu<<<num_thread_blocks, threadsPerBlock>>>(src_gpu,strongestDst_gpu, strongNeighbor_gpu, numEdges);
		temp1_gpu = strongestDst_gpu;
		strongestDst_gpu = NULL;
		
		//reminder: expected first iteration strongNeighbor_gpu: 3 4 5 4 1 2 3 4 7
		
        //Step 2: Each vertex checks if there is a handshaking
        check_handshaking_gpu<<<num_thread_blocks, threadsPerBlock>>>(strongNeighbor_gpu, matches_gpu, numVertices);
		
		//Step 3: filter
		
		//Step 3a: decide which edges to keep (marked with a 1) and filter (marked with a 0)
		int * keepEdges_gpu = temp1_gpu;
		temp1_gpu = NULL;
		markFilterEdges_gpu<<<num_thread_blocks, threadsPerBlock>>>(src_gpu, dst_gpu, matches_gpu, keepEdges_gpu, numEdges);
		
		
		//Step 3b: get new indices in edge list for the edges we're going to keep
		int * newEdgeLocs_gpu = keepEdges_gpu;
		keepEdges_gpu = NULL;
		for(int distance = 0; distance <= numEdges; distance = max(1, distance * 2)) {
			exclusive_prefix_sum_gpu<<<num_thread_blocks, threadsPerBlock>>>(newEdgeLocs_gpu, temp2_gpu, distance, numEdges+1);
			swapArray((void**) &newEdgeLocs_gpu, (void**) &temp2_gpu);
		}
		
		//note: temp1 is still in use, until we're done with newEdgeLocs_gpu
		
		//Step 3c: check if we're done matching
		int lastLoc = 0;
		hipMemcpy(&lastLoc, &(newEdgeLocs_gpu[numEdges]), sizeof(int), hipMemcpyDeviceToHost);
		if(lastLoc < 2) {
			//termination: fewer than two nodes remain unmatched
			break;
		} else if(lastLoc == numEdges) {
			//termination: no additional matches are possible
			break;
		}
		
		//Step 3d: pack the src, dst, and weight arrays in accordance with new edge locations
		packGraph_gpu<<<num_thread_blocks, threadsPerBlock>>>(temp2_gpu, src_gpu, temp3_gpu, dst_gpu, temp4_gpu, weight_gpu, newEdgeLocs_gpu, numEdges);
		swapArray((void**) &temp2_gpu, (void**) &src_gpu);
		swapArray((void**) &temp3_gpu, (void**) &dst_gpu);
		swapArray((void**) &temp4_gpu, (void**) &weight_gpu);
		
		temp1_gpu = newEdgeLocs_gpu;
		newEdgeLocs_gpu = NULL;
		
		//note: we're done with the current contents of all the temporary arrays
		
		//Set new number of edges:
		numEdges = lastLoc;
		
		if(iter > numVertices) {
			cerr << "Error: matching has been running too long; breaking loop now\n";
			break;
		}
    }
	
	hipMemcpy(matches, matches_gpu, numVertices * sizeof(int), hipMemcpyDeviceToHost);
	
	//Wait until pending GPU operations are complete:
	hipDeviceSynchronize();
	
	//free GPU arrays
	hipFree(strongNeighbor_gpu);
	hipFree(matches_gpu);
	hipFree(src_gpu);
	hipFree(dst_gpu);
	hipFree(weight_gpu);
	hipFree(temp1_gpu);
	hipFree(temp2_gpu);
	hipFree(temp3_gpu);
	hipFree(temp4_gpu);
	
	hipError_t hipError_t;
	hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess) {
		cerr << "Warning: one or more CUDA errors occurred. Try using cuda-gdb to debug. Error message: \n\t" <<hipGetErrorString(hipError_t) << "\n";
	}
	
	return iter + 1;
}

void one_way_handshake_wrapper(GraphData graph, int *& matches, int numthreads)
{
	fprintf(stderr, "Start One Way Matching ... \n");

    struct timeval beginTime, endTime;

    setTime(&beginTime);

	int iter = one_way_handshake(graph, matches, numthreads);

    setTime(&endTime);

    fprintf(stderr, "Done matching.\n");

    fprintf(stderr, "Performed matching for %ld iterations\n", iter);
    fprintf(stderr, "One Way Handshaking Matching Time: %.2f ms\n",
            getTime(&beginTime, &endTime));
}
