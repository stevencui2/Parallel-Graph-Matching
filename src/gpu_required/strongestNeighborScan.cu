#include "hip/hip_runtime.h"
/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */
#include <stdio.h>
#include <stdlib.h>

int getStrongerThreadIndex(int * oldWeight, int * oldDst, int curTID, int compareTID) {
    int strongerIndex;
    const int COMPARE_T_WEIGHT = oldWeight[compareTID];
    const int CUR_T_WEIGHT = oldWeight[curTID];

    if(COMPARE_T_WEIGHT > CUR_T_WEIGHT) {
        strongerIndex = compareTID;
    }
    else if(COMPARE_T_WEIGHT < CUR_T_WEIGHT) {
        strongerIndex = curTID;
    }
    // case: equal weights, take node with smaller vID
    else {
        const int COMPARE_T_D = oldDst[compareTID];
        const int CUR_T_D = oldDst[curTID];

        if(COMPARE_T_D < CUR_T_D) {
            strongerIndex = compareTID;
        } else {
            strongerIndex = curTID;
        };
    }

    return strongerIndex;
}

__global__ void strongestNeighborScan_gpu(
        int * src,
        int * oldDst, int * newDst,
        int * oldWeight, int * newWeight,
        int * madeChanges,
        int distance,
        int numEdges
        ) {
    // Calculate thread work
    const int NUM_THREADS = blockDim.x * gridDim.x;
    const int THREAD_WORK = numEdges / NUM_THREADS;

    //Get thread ID
    const int FIRST_T_ID = blockIdx.x * blockDim.x + threadIdx.x;

    for(const int CUR_T_ID = FIRST_T_ID; CUR_T_ID <= numEdges; CUR_T_ID += NUM_THREADS) {
        // get compare thread index, enforce 0 bound
        const int COMPARE_T_ID = CUR_T_ID - distance > 0 ? CUR_T_ID - distance : 0;

        // case : shared segment
        if( src[COMPARE_T_ID] == src[CUR_T_ID]) {
            const int STRONGER_INDEX = getStrongerThreadIndex(oldWeight, oldDst, CUR_T_ID, COMPARE_T_ID);
            //Set new destination
            newDst[CUR_T_ID] = oldDst[STRONGER_INDEX];

            //Set new weight
            newWeight[CUR_T_ID] = oldWeight[STRONGER_INDEX];

            //Flag any changes
            if(newDst[CUR_T_ID] != oldDst[CUR_T_ID]) { *madeChanges = 1; };
        }
        // case : different segment
        else {
            // defaults to no change
            newDst[CUR_T_ID] = oldDst[CUR_T_ID];
            newWeight[CUR_T_ID] = oldWeight[CUR_T_ID];
        }
    }
}
