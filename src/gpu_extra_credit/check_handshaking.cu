
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void check_handshaking_gpu(int * strongNeighbor, int * matches, int numNodes) {
	// Get Thread ID
	const int NUM_THREADS = blockDim.x * gridDim.x;
	const int COL = blockIdx.x * blockDim.x + threadIdx.x;
	const int ROW = blockIdx.y * blockDim.y + threadIdx.y;
	const int FIRST_T_ID = COL + ROW * NUM_THREADS;

	for(int curTID = FIRST_T_ID; curTID <= numNodes; curTID += NUM_THREADS) {
		if(strongNeighbor[curTID] == strongNeighbor[curTID+1] && matches[curTID] == -1){
			matches[curTID] = strongNeighbor[curTID+1];
		}
	}
}
