
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


/*
strong neighbors
5
7
8
4
3
0
7
1
2
1
*/
__global__ void check_handshaking_gpu(int * strongNeighbor, int * matches, int numNodes) {
	// Get Thread ID
	const int NUM_THREADS = blockDim.x * gridDim.x;
	const int COL = blockIdx.x * blockDim.x + threadIdx.x;
	const int ROW = blockIdx.y * blockDim.y + threadIdx.y;
	const int FIRST_T_ID = COL + ROW * NUM_THREADS;

	for(int curTID = FIRST_T_ID; curTID <= numNodes; curTID += NUM_THREADS) {
		if(matches[curTID] == -1) {
			if(curTID == strongNeighbor[strongNeighbor[curTID]]) {
				matches[curTID] = strongNeighbor[curTID];
			}
		}
	}
}
